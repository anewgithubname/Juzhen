/**
 * @file demo_mnist.cu
 * @brief MNIST multiclass logistic classifier
 * @author Song Liu (song.liu@bristol.ac.uk)
 *
 * This file contains all essential matrix operations.
 * Whatever you do, please keep it as simple as possible.
 *
    Copyright (C) 2022 Song Liu (song.liu@bristol.ac.uk)

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.

 */

// #define CPU_ONLY

#include "../ml/layer.hpp"
#include "../cpp/juzhen.hpp"
#include <math.h>
#include <ctime>
#include <thread>

using namespace std;
using namespace Juzhen;

std::string getCPUInfo();
std::string getGPUInfo();
std::string getRAMInfo();

void send_computing_time(double elapsed){

    std::cout << "Elapsed time: " << elapsed << " ms" << std::endl;

    std::string CPU = getCPUInfo(), GPU = getGPUInfo(), MEM = getRAMInfo();
    std::string jsonData = "{\\\"cpu\\\":\\\"" + CPU + "\\\", \\\"gpu\\\":\\\"" + 
                GPU + "\\\", \\\"ram\\\":\\\"" + MEM + "\\\", \\\"time\\\":\\\"" + 
                std::to_string(elapsed) + "\\\"}";

    std::string server = BENCHMARKSERVER;
    // Prepare the curl command. Notice the escape of double quotes for Windows command line.
    std::string command = "curl -X POST -H \"Content-Type: application/json\" -d \"" 
                            + jsonData + "\" http://" + server +":5000/update";

    std::cout << command << std::endl;
    std::cout << "See your ranking at: http://" + server + ".com:8080" << std::endl;
    // Execute the curl command using system()
    int result = system(command.c_str());

    // Check the result of the system call
    if (result != 0) {
        // Handle the error case
        ERROR_OUT;
    }
}

#ifndef CPU_ONLY
#define FLOAT CUDAfloat
inline Matrix<CUDAfloat> randn(int m, int n) { return Matrix<CUDAfloat>::randn(m, n); }
inline Matrix<CUDAfloat> ones(int m, int n) { return Matrix<CUDAfloat>::ones(m, n); }
#else
#define FLOAT float
inline Matrix<float> randn(int m, int n) { return Matrix<float>::randn(m, n); }
inline Matrix<float> ones(int m, int n) { return Matrix<float>::ones(m, n); }
#endif
#define MatrixI Matrix<int> 

// convert label Y matrix (1 X n) to one-hot encoding. 
Matrix<float> one_hot(const MatrixI& Y, int k) {
    Matrix<float> Y_one_hot("One_hot", k, Y.num_col());
    Y_one_hot.zeros();

    for (int i = 0; i < Y.num_col(); i++) {
        Y_one_hot.elem(Y.elem(0, i), i) = 1.0;
    }

    return Y_one_hot;
}

vector<Matrix<float>> mnist_dataset(){
    const int k = 10;
    std::string base = PROJECT_DIR + std::string("/datasets/MNIST");

    // check if *.matrix files exist
    FILE *fp = fopen((base + "/X.matrix").c_str(), "r");
    if (!fp) {
        // unzip dataset.zip to the folder 
        std::string command = "unzip " + base + "/dataset.zip -d " + base;
        int result = system(command.c_str());
        if (result != 0) {
            ERROR_OUT;
        }
    }


    auto X = read<float>(base + "/X.matrix"); 
    std::cout << "size of X: " << X.num_row() << " " << X.num_col() << std::endl;

    auto labels = read<int>(base +"/Y.matrix"); 
    std::cout << "size of labels: " << labels.num_row() << " " << labels.num_col() << std::endl;

    auto Y = one_hot(labels, k);
    std::cout << "size of Y: " << Y.num_row() << " " << Y.num_col() << std::endl;

    auto Xt = read<float>(base + "/T.matrix");
    std::cout << "size of Xt: " << Xt.num_row() << " " << Xt.num_col() << std::endl;

    auto labels_t = read<int>(base + "/YT.matrix"); 
    std::cout << "size of labels_t: " << labels_t.num_row() << " " << labels_t.num_col() << std::endl;

    auto Yt = one_hot(labels_t, k);
    std::cout << "size of Yt: " << Yt.num_row() << " " << Yt.num_col() << std::endl;

    return {X, Y, Xt, Yt};
}

int compute() {
    auto t1 = std::chrono::high_resolution_clock::now();
    // spdlog::set_level(spdlog::level::debug);
#ifndef CPU_ONLY
    GPUSampler sampler(1);
#endif
    const int d = 28*28, k = 10, batchsize = 32;
    auto vecXY = mnist_dataset();
    auto X = vecXY[0]; 
    auto Y = vecXY[1];
    
    const int numbatches = X.num_col() / batchsize;

#ifndef CPU_ONLY
    auto XT = Matrix<CUDAfloat>(vecXY[2]);
#else
    auto &XT = vecXY[2];
#endif

#ifndef CPU_ONLY
    auto YT = Matrix<CUDAfloat>(vecXY[3]);
#else
    auto &YT = vecXY[3];
#endif

    // define layers
    ReluLayer<FLOAT> L0(1024, d, batchsize), L1(128, 1024, batchsize);
    LinearLayer<FLOAT> L2(k, 128, batchsize);
    // logistic loss
    ZeroOneLayer<FLOAT> L3t(XT.num_col(), YT);

    // nns are linked lists containing layers
    list<Layer<FLOAT>*> trainnn({ &L2, &L1, &L0 }), testnn({ &L3t, &L2, &L1, &L0 });

    // //if file exists, load weights
    // FILE *fp = fopen((std::string(PROJECT_DIR) + "/mnist.weights").c_str(), "r");
    // if (fp) {
    //     fclose(fp);
    //     loadweights(trainnn, std::string(PROJECT_DIR) + "/mnist.weights");
    // }

    // sgd
    int iter = 0;
    while (iter < 10000) {
        int batch_id = (iter % numbatches);

        // obtaining batches
#ifndef CPU_ONLY
        auto X_i = Matrix<FLOAT>(X.columns(batchsize * batch_id, batchsize * (batch_id + 1)));
        auto Y_i = Matrix<FLOAT>(Y.columns(batchsize * batch_id, batchsize * (batch_id + 1)));
#else
        auto X_i = X.columns(batchsize * batch_id, batchsize * (batch_id + 1));
        auto Y_i = Y.columns(batchsize * batch_id, batchsize * (batch_id + 1));
#endif

        // forward-backward pass
        forward(trainnn, X_i);
        LogisticLayer<FLOAT> L3(batchsize, Y_i);
        trainnn.push_front(&L3);

        backprop(trainnn, X_i);
        trainnn.pop_front();
        if (iter % 1000 == 0) {
#ifndef CPU_ONLY
            cout << "Misclassification Rate: " << forward(testnn, XT).to_host().elem(0, 0) << endl;
#else
            cout << "Misclassification Rate: " << forward(testnn, XT).elem(0, 0) << endl;
#endif
        }

        iter++;
    }

    // dumpweights(trainnn, std::string(PROJECT_DIR) + "/mnist.weights");

    auto t2 = std::chrono::high_resolution_clock::now();
    auto elapsed = time_in_ms(t1, t2);

    send_computing_time(elapsed);

    return 0;
}